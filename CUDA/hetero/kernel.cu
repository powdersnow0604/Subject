#include "hip/hip_runtime.h"
#include "kernelCall.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void kernel(void)
{
	printf("Device code running \
	on the GPU\n");
}

void kernelCall(void)
{
	kernel <<<1, 10>>> ();
	hipDeviceSynchronize();
}