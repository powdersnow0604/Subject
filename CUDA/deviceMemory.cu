#include "hip/hip_runtime.h"

#include <stdio.h>

void checkDeviceMemory(void)
{
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Device memory (free/total) = %zu/%zu bytes\n", free, total);
}


int main(void)
{
    int *dPtr;
    hipError_t errorCode;

    checkDeviceMemory();
    errorCode = hipMalloc(&dPtr, sizeof(int) << 20);
    printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
    checkDeviceMemory();

    errorCode = hipMemset(dPtr, 0, sizeof(int) <<  20);
    printf("hipMemset - %s\n", hipGetErrorName(errorCode));

    errorCode = hipFree(dPtr);
    printf("hipFree - %s\n", hipGetErrorName(errorCode));
    checkDeviceMemory();

    return 0;
}