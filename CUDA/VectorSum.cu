#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "Profiler.h"

// The size of the vector
#define NUM_DATA 1048576

// Simple vector sum kernel (Max vector size : 1024)
__global__ void vecAdd(int *_a, int *_b, int *_c)
{
	int tID = threadIdx.x + blockDim.x * blockIdx.x;
	if(tID < NUM_DATA) _c[tID] = _a[tID] + _b[tID];
}

using namespace std::chrono;

int main(void)
{
	int *a, *b, *c, *hc; // Vectors on the host
	int *da, *db, *dc;	 // Vectors on the device

	Profiler sec_total("CUDA total");
	Profiler sec_kernel("Computation(Kernel)");
	Profiler sec_transHD("Data Trans.(Hoat -> Device)");
	Profiler sec_transDH("CUDA total.(Device -> Host)");
	Profiler sec_vecaddH("VecAdd on Host");

	int memSize = sizeof(int) * NUM_DATA;
	printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

	// Memory allocation on the host-side
	a = new int[NUM_DATA];
	memset(a, 0, memSize);
	b = new int[NUM_DATA];
	memset(b, 0, memSize);
	c = new int[NUM_DATA];
	memset(c, 0, memSize);
	hc = new int[NUM_DATA];
	memset(hc, 0, memSize);

	// Data generation
	for (int i = 0; i < NUM_DATA; i++)
	{
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	// Vector sum on host (for performance comparision)
	{
		auto _ = sec_vecaddH.profile();

		for (int i = 0; i < NUM_DATA; i++)
			hc[i] = a[i] + b[i];
	}

	// Memory allocation on the device-side
	hipMalloc(&da, memSize);
	hipMemset(da, 0, memSize);
	hipMalloc(&db, memSize);
	hipMemset(db, 0, memSize);
	hipMalloc(&dc, memSize);
	hipMemset(dc, 0, memSize);

	{
		auto P1 = sec_total.profile();

		// Data copy : Host -> Device
		{
			auto _ = sec_transHD.profile();
			hipMemcpy(da, a, memSize, hipMemcpyHostToDevice);
			hipMemcpy(db, b, memSize, hipMemcpyHostToDevice);
		}

		// Kernel call
		{
			auto _ = sec_kernel.profile();
			vecAdd<<<(uint64_t)ceil(NUM_DATA/1024.), 1024>>>(da, db, dc);
			hipDeviceSynchronize();
		}

		// Copy results : Device -> Host
		{
			auto _ = sec_transDH.profile();
			hipMemcpy(c, dc, memSize, hipMemcpyDeviceToHost);
		}
	}
	
	// Release device memory
	hipFree(da);
	hipFree(db);
	hipFree(dc);

	// Check results
	bool result = true;
	for (int i = 0; i < NUM_DATA; i++)
	{
		if (hc[i] != c[i])
		{
			printf("[%d] The result is not matched! (%d, %d)\n", i, hc[i], c[i]);
			result = false;
		}
	}

	if (result)
		printf("GPU works well!\n");

	sec_total.report<microseconds>();
	sec_kernel.report<microseconds>();
	sec_transHD.report<microseconds>();
	sec_transDH.report<microseconds>();
	sec_vecaddH.report<microseconds>();

	// Release host memory
	delete[] a;
	delete[] b;
	delete[] c;

	return 0;
}